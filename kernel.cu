﻿
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include <stdio.h>
//
//cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    cudaError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // cudaDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = cudaDeviceReset();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//cudaError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    cudaError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = cudaSetDevice(0);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = cudaMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = cudaMemcpy(dev_a, a, size * sizeof(int), cudaMemcpyHostToDevice);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = cudaMemcpy(dev_b, b, size * sizeof(int), cudaMemcpyHostToDevice);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = cudaGetLastError();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // cudaDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = cudaDeviceSynchronize();
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = cudaMemcpy(c, dev_c, size * sizeof(int), cudaMemcpyDeviceToHost);
//    if (cudaStatus != cudaSuccess) {
//        fprintf(stderr, "cudaMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    cudaFree(dev_c);
//    cudaFree(dev_a);
//    cudaFree(dev_b);
//    
//    return cudaStatus;
//}


//#include<cuda_runtime.h>
//#include <cusparse.h>
//#include<cublas_v2.h>
//#include<iostream>

//int main() {
//    // 初始化 CUSPARSE
//    cusparseHandle_t cusparseHandle;
//    cusparseCreate(&cusparseHandle);
//
//    // 定义稀疏矩阵 (CSR 格式)
//    int rows = 3; // 矩阵的行数
//    int cols = 3; // 矩阵的列数
//    int nnz = 4;  // 非零元素的数量
//
//    // CSR 格式的数据
//    float h_values[] = { 1.0f, 2.0f, 3.0f, 4.0f };        // 非零元素的值
//    int h_columns[] = { 0, 1, 1, 2 };                     // 非零元素对应的列索引
//    int h_rowPointers[] = { 0, 1, 3, 4 };                 // 每行非零元素的起始索引
//
//    // 向量
//    float h_x[] = { 1.0f, 2.0f, 3.0f };                   // 输入向量
//    float h_y[3] = { 0.0f, 0.0f, 0.0f };                  // 输出向量初始化为 0
//
//    // 在设备上分配内存
//    float* d_values, * d_x, * d_y;
//    int* d_columns, * d_rowPointers;
//
//    cudaMalloc((void**)&d_values, nnz * sizeof(float));
//    cudaMalloc((void**)&d_columns, nnz * sizeof(int));
//    cudaMalloc((void**)&d_rowPointers, (rows + 1) * sizeof(int));
//    cudaMalloc((void**)&d_x, cols * sizeof(float));
//    cudaMalloc((void**)&d_y, rows * sizeof(float));
//
//    // 将数据从主机复制到设备
//    cudaMemcpy(d_values, h_values, nnz * sizeof(float), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_columns, h_columns, nnz * sizeof(int), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_rowPointers, h_rowPointers, (rows + 1) * sizeof(int), cudaMemcpyHostToDevice);
//    cudaMemcpy(d_x, h_x, cols * sizeof(float), cudaMemcpyHostToDevice);
//
//    // 定义 alpha 和 beta
//    float alpha = 1.0f;
//    float beta = 0.0f;
//
//    // 创建稀疏矩阵描述符 (cusparseSpMatDescr_t)
//    cusparseSpMatDescr_t matA;
//    cusparseCreateCsr(&matA, rows, cols, nnz,
//        d_rowPointers, d_columns, d_values,
//        CUSPARSE_INDEX_32I, CUSPARSE_INDEX_32I,
//        CUSPARSE_INDEX_BASE_ZERO, CUDA_R_32F);
//
//    // 创建向量描述符 (cusparseDnVecDescr_t)
//    cusparseDnVecDescr_t vecX, vecY;
//    cusparseCreateDnVec(&vecX, cols, d_x, CUDA_R_32F);
//    cusparseCreateDnVec(&vecY, rows, d_y, CUDA_R_32F);
//
//    // 分配临时缓冲区
//    size_t bufferSize = 0;
//    void* dBuffer = nullptr;
//    cusparseSpMV_bufferSize(cusparseHandle, CUSPARSE_OPERATION_NON_TRANSPOSE,
//        &alpha, matA, vecX, &beta, vecY,
//        CUDA_R_32F, CUSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
//    cudaMalloc(&dBuffer, bufferSize);
//
//    // 执行稀疏矩阵-向量乘法 (SpMV)
//    cusparseSpMV(cusparseHandle, CUSPARSE_OPERATION_NON_TRANSPOSE,
//        &alpha, matA, vecX, &beta, vecY,
//        CUDA_R_32F, CUSPARSE_SPMV_ALG_DEFAULT, dBuffer);
//
//    // 将结果从设备复制到主机
//    cudaMemcpy(h_y, d_y, rows * sizeof(float), cudaMemcpyDeviceToHost);
//
//    // 打印结果
//    std::cout << "Result vector y:" << std::endl;
//    for (int i = 0; i < rows; i++) {
//        std::cout << h_y[i] << " ";
//    }
//    std::cout << std::endl;
//
//    // 释放资源
//    cusparseDestroySpMat(matA);
//    cusparseDestroyDnVec(vecX);
//    cusparseDestroyDnVec(vecY);
//    cudaFree(dBuffer);
//    cudaFree(d_values);
//    cudaFree(d_columns);
//    cudaFree(d_rowPointers);
//    cudaFree(d_x);
//    cudaFree(d_y);
//    cusparseDestroy(cusparseHandle);
//
//    return 0;
//}





#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>

int main() {
    hipsparseHandle_t handle;
    hipsparseStatus_t status = hipsparseCreate(&handle);

    if (status != HIPSPARSE_STATUS_SUCCESS) {
        std::cerr << "CUSPARSE 初始化失败！" << std::endl;
        return -1;
    }

    std::cout << "CUSPARSE 初始化成功！" << std::endl;
    hipsparseDestroy(handle);
    return 0;
}
