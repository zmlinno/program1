#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>



//// CUDA内核：稀疏矩阵加法（COO格式）
//__global__ void sparseMatrixAddKernel(int* row_a, int* col_a, int* val_a, int nonZeroA,
//    int* row_b, int* col_b, int* val_b, int nonZeroB,
//    int* row_c, int* col_c, int* val_c, int* nonZeroC) {
//    int i = threadIdx.x; // 线程索引
//
//    // 检查当前线程是否在合法范围内
//    if (i < nonZeroA) {
//        for (int j = 0; j < nonZeroB; j++) {
//            if (row_a[i] == row_b[j] && col_a[i] == col_b[j]) {
//                // 位置相同，执行加法
//                val_c[i] = val_a[i] + val_b[j];
//                row_c[i] = row_a[i];
//                col_c[i] = col_a[i];
//                atomicAdd(nonZeroC, 1); // 使用原子操作计数非零元素个数
//            }
//        }
//    }
//}
//
////运行COO格式的稀疏矩阵加法
//void runCOO()
//{
//    //模拟1000*1000的稀疏矩阵，每个矩阵有5000个非0元素
//    const int nonZeroA = 5000, nonZeroB = 5000;
//    int* row_a = (int*)malloc(nonZeroA * sizeof(int));
//    int* col_a = (int*)malloc(nonZeroA * sizeof(int));
//    int* val_a = (int*)malloc(nonZeroA * sizeof(int));
//
//    int* row_b = (int*)malloc(nonZeroB * sizeof(int));
//    int* col_b = (int*)malloc(nonZeroB * sizeof(int));
//    int* val_b = (int*)malloc(nonZeroB * sizeof(int));
//
//    int* row_c = (int*)malloc(nonZeroA * sizeof(int));
//    int* col_c = (int*)malloc(nonZeroA * sizeof(int));
//    int* val_c = (int*)malloc(nonZeroA * sizeof(int));
//
//    int nonZeroC = 0;
//
//    //初始化A和B稀疏矩阵的元素
//    for (int i = 0; i < nonZeroA; i++)
//    {
//        row_a[i] = i % 1000;
//        col_a[i] = i % 1000;
//        val_a[i] = i % 1000;
//    }
//
//    for (int i = 0; i < nonZeroB; i++)
//    {
//        row_b[i] = i % 1000;
//        col_b[i] = i % 1000;
//        val_b[i] = i % 1000;
//    }
//
//    hipError_t cudaStatus = addWithCudaCOO(row_c, col_c, row_a, col_a, val_a, nonZeroA,
//        row_b, col_b, val_b, nonZeroB, nonZeroC);
//
//    if (cudaStatus != hipSuccess)
//    {
//        fprintf(stderr, "addWithCuda failed");
//        return;
//    }
//}
//
//// 主机函数：调用CUDA内核
//hipError_t addWithCuda(int* row_c, int* col_c, int* val_c,
//    const int* row_a, const int* col_a, const int* val_a, int nonZeroA,
//    const int* row_b, const int* col_b, const int* val_b, int nonZeroB, int* nonZeroC) {
//    int* dev_row_a = 0, * dev_col_a = 0, * dev_val_a = 0;
//    int* dev_row_b = 0, * dev_col_b = 0, * dev_val_b = 0;
//    int* dev_row_c = 0, * dev_col_c = 0, * dev_val_c = 0;
//    int* dev_nonZeroC = 0;
//    hipError_t cudaStatus;
//
//    // 选择CUDA设备
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
//        goto Error;
//    }
//
//    // 分配GPU内存
//    hipMalloc((void**)&dev_row_a, nonZeroA * sizeof(int));
//    hipMalloc((void**)&dev_col_a, nonZeroA * sizeof(int));
//    hipMalloc((void**)&dev_val_a, nonZeroA * sizeof(int));
//
//    hipMalloc((void**)&dev_row_b, nonZeroB * sizeof(int));
//    hipMalloc((void**)&dev_col_b, nonZeroB * sizeof(int));
//    hipMalloc((void**)&dev_val_b, nonZeroB * sizeof(int));
//
//    hipMalloc((void**)&dev_row_c, nonZeroA * sizeof(int)); // 假设C最多有nonZeroA个非零元素
//    hipMalloc((void**)&dev_col_c, nonZeroA * sizeof(int));
//    hipMalloc((void**)&dev_val_c, nonZeroA * sizeof(int));
//    hipMalloc((void**)&dev_nonZeroC, sizeof(int));
//
//    // 将稀疏矩阵A和B的数据从主机复制到设备
//    hipMemcpy(dev_row_a, row_a, nonZeroA * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(dev_col_a, col_a, nonZeroA * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(dev_val_a, val_a, nonZeroA * sizeof(int), hipMemcpyHostToDevice);
//
//    hipMemcpy(dev_row_b, row_b, nonZeroB * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(dev_col_b, col_b, nonZeroB * sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(dev_val_b, val_b, nonZeroB * sizeof(int), hipMemcpyHostToDevice);
//
//    // 初始化非零元素计数器
//    hipMemset(dev_nonZeroC, 0, sizeof(int));
//
//    // 启动CUDA内核
//    sparseMatrixAddKernel << <1, nonZeroA >> > (dev_row_a, dev_col_a, dev_val_a, nonZeroA,
//        dev_row_b, dev_col_b, dev_val_b, nonZeroB,
//        dev_row_c, dev_col_c, dev_val_c, dev_nonZeroC);
//
//    // 检查内核启动的错误
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//
//    // 同步设备
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // 将结果从设备复制回主机
//    hipMemcpy(row_c, dev_row_c, nonZeroA * sizeof(int), hipMemcpyDeviceToHost);
//    hipMemcpy(col_c, dev_col_c, nonZeroA * sizeof(int), hipMemcpyDeviceToHost);
//    hipMemcpy(val_c, dev_val_c, nonZeroA * sizeof(int), hipMemcpyDeviceToHost);
//    hipMemcpy(nonZeroC, dev_nonZeroC, sizeof(int), hipMemcpyDeviceToHost);
//
//Error:
//    hipFree(dev_row_a);
//    hipFree(dev_col_a);
//    hipFree(dev_val_a);
//    hipFree(dev_row_b);
//    hipFree(dev_col_b);
//    hipFree(dev_val_b);
//    hipFree(dev_row_c);
//    hipFree(dev_col_c);
//    hipFree(dev_val_c);
//    hipFree(dev_nonZeroC);
//
//    return cudaStatus;
//}
//
//// 主函数
//int main() {
//    const int nonZeroA = 3, nonZeroB = 3;
//    int row_a[nonZeroA] = { 0, 1, 2 };
//    int col_a[nonZeroA] = { 0, 1, 2 };
//    int val_a[nonZeroA] = { 10, 20, 30 };
//
//    int row_b[nonZeroB] = { 0, 1, 2 };
//    int col_b[nonZeroB] = { 0, 1, 2 };
//    int val_b[nonZeroB] = { 5, 15, 25 };
//
//    int row_c[nonZeroA], col_c[nonZeroA], val_c[nonZeroA];
//    int nonZeroC = 0;
//
//    // 调用CUDA加法函数
//    hipError_t cudaStatus = addWithCuda(row_c, col_c, val_c, row_a, col_a, val_a, nonZeroA,
//        row_b, col_b, val_b, nonZeroB, &nonZeroC);
//
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    // 打印结果矩阵C
//    printf("Resulting sparse matrix C (in COO format):\n");
//    for (int i = 0; i < nonZeroC; i++) {
//        printf("C(%d, %d) = %d\n", row_c[i], col_c[i], val_c[i]);
//    }
//
//    return 0;
//}


// CUDA内核：稀疏矩阵加法（CSR格式）
__global__ void sparseMatrixAddKernelCSR(int* row_ptr_a, int* col_a, int* val_a, int nonZeroA,
    int* row_ptr_b, int* col_b, int* val_b, int nonZeroB,
    int* row_ptr_c, int* col_c, int* val_c, int* nonZeroC) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    // 确保线程在合法的范围内
    if (row < 1000) {
        int startA = row_ptr_a[row];
        int endA = row_ptr_a[row + 1];

        int startB = row_ptr_b[row];
        int endB = row_ptr_b[row + 1];

        int idxC = row_ptr_c[row];

        // 遍历A和B的非零元素并执行加法
        while (startA < endA && startB < endB) {
            if (col_a[startA] == col_b[startB]) {
                val_c[idxC] = val_a[startA] + val_b[startB];
                col_c[idxC] = col_a[startA];
                startA++;
                startB++;
            }
            else if (col_a[startA] < col_b[startB]) {
                val_c[idxC] = val_a[startA];
                col_c[idxC] = col_a[startA];
                startA++;
            }
            else {
                val_c[idxC] = val_b[startB];
                col_c[idxC] = col_b[startB];
                startB++;
            }
            idxC++;
        }

        // 将剩余的非零元素加到C中
        while (startA < endA) {
            val_c[idxC] = val_a[startA];
            col_c[idxC] = col_a[startA];
            startA++;
            idxC++;
        }

        while (startB < endB) {
            val_c[idxC] = val_b[startB];
            col_c[idxC] = col_b[startB];
            startB++;
            idxC++;
        }

        row_ptr_c[row + 1] = idxC; // 更新行指针
    }
}

// 主机函数：调用CUDA内核（CSR格式）
hipError_t addWithCudaCSR(int* row_ptr_c, int* col_c, int* val_c,
    const int* row_ptr_a, const int* col_a, const int* val_a, int nonZeroA,
    const int* row_ptr_b, const int* col_b, const int* val_b, int nonZeroB, int* nonZeroC) {
    int* dev_row_ptr_a = 0, * dev_col_a = 0, * dev_val_a = 0;
    int* dev_row_ptr_b = 0, * dev_col_b = 0, * dev_val_b = 0;
    int* dev_row_ptr_c = 0, * dev_col_c = 0, * dev_val_c = 0;
    int* dev_nonZeroC = 0;
    hipError_t cudaStatus;

    // 分配GPU内存
    hipMalloc((void**)&dev_row_ptr_a, 1001 * sizeof(int));
    hipMalloc((void**)&dev_col_a, nonZeroA * sizeof(int));
    hipMalloc((void**)&dev_val_a, nonZeroA * sizeof(int));

    hipMalloc((void**)&dev_row_ptr_b, 1001 * sizeof(int));
    hipMalloc((void**)&dev_col_b, nonZeroB * sizeof(int));
    hipMalloc((void**)&dev_val_b, nonZeroB * sizeof(int));

    hipMalloc((void**)&dev_row_ptr_c, 1001 * sizeof(int));
    hipMalloc((void**)&dev_col_c, nonZeroA * sizeof(int));
    hipMalloc((void**)&dev_val_c, nonZeroA * sizeof(int));

    // 将稀疏矩阵A和B的数据从主机复制到设备
    hipMemcpy(dev_row_ptr_a, row_ptr_a, 1001 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_a, col_a, nonZeroA * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_val_a, val_a, nonZeroA * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(dev_row_ptr_b, row_ptr_b, 1001 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_col_b, col_b, nonZeroB * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_val_b, val_b, nonZeroB * sizeof(int), hipMemcpyHostToDevice);

    // 启动CUDA内核
    int blocks = (1000 + BLOCK_SIZE - 1) / BLOCK_SIZE;
    sparseMatrixAddKernelCSR << <blocks, BLOCK_SIZE >> > (dev_row_ptr_a, dev_col_a, dev_val_a, nonZeroA,
        dev_row_ptr_b, dev_col_b, dev_val_b, nonZeroB,
        dev_row_ptr_c, dev_col_c, dev_val_c, dev_nonZeroC);

    // 检查内核启动的错误
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSR kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // 同步设备
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
        goto Error;
    }

    // 将结果从设备复制回主机
    hipMemcpy(row_ptr_c, dev_row_ptr_c, 1001 * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(col_c, dev_col_c, nonZeroA * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(val_c, dev_val_c, nonZeroA * sizeof(int), hipMemcpyDeviceToHost);

Error:
    hipFree(dev_row_ptr_a);
    hipFree(dev_col_a);
    hipFree(dev_val_a);
    hipFree(dev_row_ptr_b);
    hipFree(dev_col_b);
    hipFree(dev_val_b);
    hipFree(dev_row_ptr_c);
    hipFree(dev_col_c);
    hipFree(dev_val_c);

    return cudaStatus;
}

// 主函数：运行CSR稀疏矩阵加法
void runCSR() {
    const int nonZeroA = 5000, nonZeroB = 5000;
    int row_ptr_a[1001] = { 0 }, row_ptr_b[1001] = { 0 }, row_ptr_c[1001] = { 0 };
    int* col_a = (int*)malloc(nonZeroA * sizeof(int));
    int* val_a = (int*)malloc(nonZeroA * sizeof(int));
    int* col_b = (int*)malloc(nonZeroB * sizeof(int));
    int* val_b = (int*)malloc(nonZeroB * sizeof(int));
    int* col_c = (int*)malloc(nonZeroA * sizeof(int));
    int* val_c = (int*)malloc(nonZeroA * sizeof(int));

    // 初始化A和B稀疏矩阵的元素
    for (int i = 0; i < nonZeroA; i++) {
        col_a[i] = i % 1000;
        val_a[i] = i + 1;
    }

    for (int i = 0; i < nonZeroB; i++) {
        col_b[i] = i % 1000;
        val_b[i] = i + 2;
    }

    // 调用CUDA加法函数
    hipError_t cudaStatus = addWithCudaCSR(row_ptr_c, col_c, val_c, row_ptr_a, col_a, val_a, nonZeroA,
        row_ptr_b, col_b, val_b, nonZeroB, &nonZeroA);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCudaCSR failed!");
        return;
    }

    // 打印结果矩阵C
    printf("Resulting sparse matrix C (in CSR format):\n");
    for (int i = 0; i < row_ptr_c[1000]; i++) {
        printf("C(row %d) = %d\n", i, val_c[i]);
    }
}